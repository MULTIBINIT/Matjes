#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <cassert>

__global__ void k_time_m(
    hipfftDoubleComplex* h, 
    hipfftDoubleComplex* m,
    hipfftDoubleComplex* k, 
    int dim_mode,
    int N){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int n = (i/dim_mode)*dim_mode;
    if (i < N*dim_mode){
        h[i]=make_hipDoubleComplex(0.0,0.0);
        for (int j=0;j<dim_mode;j++){
            h[i] = hipCadd(h[i] ,hipCmul(k[i*dim_mode+j], m[n+j]));
        }
    }
}    

__global__ void add_cmplx(
    hipfftDoubleComplex* arr_sum, 
    hipfftDoubleComplex* arr_add,
    int N){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N){
        arr_sum[i]=hipCadd(arr_sum[i] ,arr_add[i]);
    }
}    



extern "C"{
void cuda_fft_calc_h(
    const int& dim_mode,
    int N_rep[3],
    hipfftDoubleReal*    &m_n,
    hipfftDoubleComplex* &m_f,
    hipfftDoubleComplex* &k_f,
    hipfftDoubleReal*    &h_n,
    hipfftDoubleComplex* &h_f,
    hipfftHandle*        &plan_fwd,
    hipfftHandle*        &plan_bwd){

    // get magnetic operator in fourier space
    hipfftResult result=hipfftExecD2Z(*plan_fwd,m_n,m_f);
    assert(result == HIPFFT_SUCCESS);

    hipError_t err=hipDeviceSynchronize();
    assert(err == hipSuccess);

    int NK_tot=N_rep[0]*N_rep[1]*N_rep[2];
    int N=NK_tot*dim_mode;

    int threadsPerBlock = std::min(256,N);
    int numBlocks= (N+ threadsPerBlock - 1) / threadsPerBlock;

    k_time_m<<<numBlocks, threadsPerBlock>>>(h_f, m_f, k_f, dim_mode, NK_tot);

    err=hipDeviceSynchronize();
    assert(err == hipSuccess);
    result=hipfftExecZ2D(*plan_bwd,h_f,h_n);
    assert(result == HIPFFT_SUCCESS);
    err=hipDeviceSynchronize();
    assert(err == hipSuccess);
}

void cuda_fft_add_cmplx(
    const int N,
    hipfftDoubleComplex* &arr_sum,
    hipfftDoubleComplex* &arr_add){

    int threadsPerBlock = std::min(256,N);
    int numBlocks= (N+ threadsPerBlock - 1) / threadsPerBlock;

    add_cmplx<<<numBlocks, threadsPerBlock>>>(arr_sum, arr_add,N);

    hipError_t err=hipDeviceSynchronize();
    assert(err == hipSuccess);
}

}
