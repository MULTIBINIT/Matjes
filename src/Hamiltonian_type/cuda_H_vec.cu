#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cassert>
#ifdef CPP_MPI
#include <mpi.h>
#endif
using namespace std;

extern "C"{


void cuda_fvec_alloccopy(
    cusparseDnVecDescr* &vec_in, 
    cusparseDnVecDescr* &vec_out){

    double *val_in, *val_out;
    //get references
    hipDataType dat_type;
    int64_t size;
    auto stat=hipsparseDnVecGet(vec_in,&size, (void**) &val_in,&dat_type);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);

    //allocate and copy new device memory 
    auto err =hipMalloc( (void**) &val_out, (size_t) sizeof(double)*size);
    assert( hipSuccess == err );
    err = hipMemcpy(val_out, val_in, (size_t) sizeof(double)*size , hipMemcpyDeviceToDevice);
    assert( hipSuccess == err);
    //initialize type
    stat= hipsparseCreateDnVec(&vec_out,  (int64_t) size, val_out,  HIP_R_64F);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    }

void cuda_fvec_init(
    cusparseDnVecDescr* &vec, 
    int size){

    double *val;
    //allocate device memory 
    auto err =hipMalloc( (void**) &val, (size_t) sizeof(double)*size);
    assert( hipSuccess == err );
    //initialize type
    auto stat= hipsparseCreateDnVec(&vec,  (int64_t) size, val,  HIP_R_64F);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    }

void cuda_fvec_destroy(
    cusparseDnVecDescr* &vec){

    double *val;
    //free device memory
    auto stat=hipsparseDnVecGetValues(vec, (void**) &val);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    auto err=hipFree(val);
    assert( hipSuccess == err);
    
    //free host memory
    stat=hipsparseDestroyDnVec(vec);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
}

void cuda_fvec_set(
    cusparseDnVecDescr* &vec,
    const double arr_in[]){

    //get references
    double *val;
    hipDataType dat_type;
    int64_t size;
    auto stat=hipsparseDnVecGet(vec,&size, (void**) &val,&dat_type);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    //copy array
    auto err = hipMemcpy(val, arr_in, (size_t) sizeof(double)*size , hipMemcpyHostToDevice);
    assert( hipSuccess == err);
}

void cuda_fvec_get(
    cusparseDnVecDescr* &vec,
    double arr_out[]){

    //get references
    double *val;
    hipDataType dat_type;
    int64_t size;
    auto stat=hipsparseDnVecGet(vec,&size, (void**) &val,&dat_type);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    //copy array
    auto err = hipMemcpy(arr_out, val, (size_t) sizeof(double)*size , hipMemcpyDeviceToHost);
    assert( hipSuccess == err);
}
}
