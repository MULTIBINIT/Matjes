#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cassert>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#ifdef CPP_MPI
#include <mpi.h>
#endif
using namespace std;

//__global__ void add_col_mat(
//    double* res, 
//    int* row,
//    int* col,
//    double* val,
//    int col_ind,
//    int ind,
//    double val_vec){
//
//    int i = blockDim.x * blockIdx.x + threadIdx.x;
//    int Nentry=row[col_ind+1]-row[col_ind];
//    int col_ind;
////    printf("Nentry: %d \n", Nentry);
//    if (i < Nentry){
//        int col_ind=row[col_ind]+i;
//        res[col_ind]=res[col_ind] + val[col_ind] * val_vec;
//    }
//}    



extern "C"{

void cuda_create_handle(
    hipsparseHandle_t* &handle
    ){
    handle = new hipsparseHandle_t;
    hipsparseStatus_t stat = hipsparseCreate(handle);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
}

void cuda_H_init(
    // initializes cuda csr 
    const int nnz,
    const int Hdim[2],
    int rows[],
    int cols[],
    double arr_in[],
    cusparseSpMatDescr*& spMatDescr,
    hipsparseHandle_t*& handle
    ){

    //define some initial helping variables
    hipsparseStatus_t stat;
    hipError_t err;
    size_t buffersize;
    int nrows=Hdim[0];
    int ncols=Hdim[1];

    //sorting the coo-matrix input into the format necessary for CUDA
      //prepare permutation array for sorting on GPU
    stat = hipsparseXcoosort_bufferSizeExt(*handle, nrows, ncols, nnz, rows, cols, &buffersize);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    hipDeviceSynchronize();
    void *pBuffer = NULL;
    hipMalloc( &pBuffer, (size_t) sizeof(char)* buffersize);
    int *P = NULL;
    hipMalloc( (void**)&P, (size_t) sizeof(int)*nnz);
    stat=hipsparseCreateIdentityPermutation(*handle, nnz, P);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    hipDeviceSynchronize();

      //Prepare actuall data-array for coo on GPU
    int* cooCols;
    int* cooRows;
    err =hipMalloc( &cooRows, (size_t) sizeof(int)*nnz);
    assert( hipSuccess == err );
    err =hipMalloc( &cooCols, (size_t) sizeof(int)*nnz);
    assert( hipSuccess == err );
    err = hipMemcpy(cooRows, rows, (size_t) sizeof(int)*nnz , hipMemcpyHostToDevice);
    assert( hipSuccess == err );
    err = hipMemcpy(cooCols, cols, (size_t) sizeof(int)*nnz , hipMemcpyHostToDevice);
    assert( hipSuccess == err );
    hipDeviceSynchronize();

      //Finally sort the cooRows and cooCols
    stat = hipsparseXcoosortByRow(*handle, nrows, ncols, nnz, cooRows, cooCols, P, pBuffer);
    assert( hipSuccess == err );
    hipDeviceSynchronize();

    //Prepare and get csrRow
    int* csrRow;
    err =hipMalloc( &csrRow, (size_t) sizeof(int)*(nrows+1));
    assert( hipSuccess == err );
    hipDeviceSynchronize();
    stat =  hipsparseXcoo2csr(*handle, cooRows, nnz, nrows, csrRow, HIPSPARSE_INDEX_BASE_ONE);
    assert( hipSuccess == err );
    hipDeviceSynchronize();

    //Move values to GPU
    double* values;
    err =hipMalloc( &values, (size_t) sizeof(double)*nnz);
    assert( hipSuccess == err );
    err = hipMemcpy(values, arr_in, (size_t) sizeof(double)*nnz , hipMemcpyHostToDevice);
    assert( hipSuccess == err );


    //Create CSR sparse matrix with Handle
    stat= hipsparseCreateCsr(&spMatDescr, nrows, ncols, nnz, csrRow, cooCols, values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_64F);
    assert( HIPSPARSE_STATUS_SUCCESS == stat);
    hipDeviceSynchronize();

    //Free stuff no longer necessary
    hipFree(pBuffer);
    hipFree(P);
    hipFree(cooRows);
}


void cuda_H_get_deviceptr(
    cusparseSpMatDescr* &mat,
    void* &csrRow,
    void* &csrCol, 
    void* &csrVal){

    //declare H_in host data
    int64_t rows, cols, nnz;
    hipsparseIndexType_t csrRowOffsetsType;
    hipsparseIndexType_t csrColIndType;
    hipsparseIndexBase_t idxBase;
    hipDataType        valueType;

    //get data from old sparse reference
    hipsparseStatus_t stat = hipsparseCsrGet(mat, &rows, &cols, &nnz
                                          ,&csrRow,&csrCol, &csrVal
                                          ,&csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
    assert( HIPSPARSE_STATUS_SUCCESS == stat);
}


void cuda_H_add(
    cusparseSpMatDescr* &mat_1,
    cusparseSpMatDescr* &mat_2,
    cusparseSpMatDescr* &mat_s,
    hipsparseHandle_t* &handle){

    //we always only want to add the arrays without prefactor
    const double alpha=1.0;
    const double beta=1.0;

    //declare types for all 3 arrays
    int64_t rows_1, rows_2;
    int64_t cols_1, cols_2;
    int64_t nnz_1, nnz_2, nnz_s;
    int    *csrRow_1=NULL, *csrRow_2=NULL, *csrRow_s=NULL;
    int    *csrCol_1=NULL, *csrCol_2=NULL, *csrCol_s=NULL;
    double *csrVal_1=NULL, *csrVal_2=NULL, *csrVal_s=NULL;
    hipsparseStatus_t stat;
    hipError_t err;

    //set size of arrays to be added
    stat= hipsparseSpMatGetSize(mat_1, &rows_1, &cols_1, &nnz_1);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );
    stat= hipsparseSpMatGetSize(mat_2, &rows_2, &cols_2, &nnz_2);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );

    //get pointers to Device data arrays of input matrices
    void *csrRow, *csrCol, *csrVal;
    cuda_H_get_deviceptr(mat_1,csrRow,csrCol,csrVal);
    csrRow_1=(int*) csrRow, csrCol_1=(int*) csrCol, csrVal_1=(double*) csrVal;
    cuda_H_get_deviceptr(mat_2,csrRow,csrCol,csrVal);
    csrRow_2=(int*) csrRow, csrCol_2=(int*) csrCol, csrVal_2=(double*) csrVal;

    //Some neccessary setting apparently
    hipsparseSetPointerMode(*handle, HIPSPARSE_POINTER_MODE_HOST);

    //Allocate sum row array
    hipMalloc((void**)&csrRow_s, sizeof(int)*(rows_1+1));

    //Set matrix descriptions for all arrays with index 1
    hipsparseMatDescr_t descr_1, descr_2, descr_s;
    stat=hipsparseCreateMatDescr(&descr_1);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );
    hipsparseSetMatIndexBase(descr_1, HIPSPARSE_INDEX_BASE_ONE);
    stat=hipsparseCreateMatDescr(&descr_2);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );
    hipsparseSetMatIndexBase(descr_2, HIPSPARSE_INDEX_BASE_ONE);
    stat=hipsparseCreateMatDescr(&descr_s);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );
    hipsparseSetMatIndexBase(descr_s, HIPSPARSE_INDEX_BASE_ONE);

    //Prepare temporary array (buffer) necessary for addition
    size_t bufferSizeInBytes;
    char *buffer = NULL;
    stat=hipsparseDcsrgeam2_bufferSizeExt(*handle, rows_1, cols_1,
        &alpha,
        descr_1, nnz_1,
        csrVal_1, csrRow_1, csrCol_1,
        &beta,
        descr_2, nnz_2,
        csrVal_2, csrRow_2, csrCol_2,
        descr_s,
        csrVal_s, csrRow_s, csrCol_s,
        &bufferSizeInBytes
        );
    assert( HIPSPARSE_STATUS_SUCCESS == stat );
    err=hipMalloc((void**)&buffer, sizeof(char)*bufferSizeInBytes);
    assert( hipSuccess == err );

    //Get nnz_s 
    int nnzC;
    int *nnzTotalDevHostPtr = &nnzC;
    stat=hipsparseXcsrgeam2Nnz(*handle, rows_1, cols_1,
        descr_1, nnz_1, csrRow_1, csrCol_1,
        descr_2, nnz_2, csrRow_2, csrCol_2,
        descr_s, csrRow_s, nnzTotalDevHostPtr,
        buffer);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );
    if (NULL != nnzTotalDevHostPtr){
        nnz_s = *nnzTotalDevHostPtr;
    }else{
        int baseC;
        hipMemcpy(&nnz_s, csrRow_s+rows_1, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, csrRow_s,        sizeof(int), hipMemcpyDeviceToHost);
        nnz_s -= baseC;
    }

    //allocate nnz_s dependent arrays (Column and value)
    hipMalloc((void**)&csrCol_s, sizeof(int)*nnz_s);
    hipMalloc((void**)&csrVal_s, sizeof(double)*nnz_s);

    //Finally get addition
    stat=hipsparseDcsrgeam2(*handle, rows_1, cols_1,
        &alpha,
        descr_1, nnz_1,
        csrVal_1, csrRow_1, csrCol_1,
        &beta,
        descr_2, nnz_2,
        csrVal_2, csrRow_2, csrCol_2,
        descr_s,
        csrVal_s, csrRow_s, csrCol_s,
        buffer);
    assert( HIPSPARSE_STATUS_SUCCESS == stat );

    //Create the Matrix handle
    stat= hipsparseCreateCsr(&mat_s, rows_1, cols_1, nnz_s, csrRow_s, csrCol_s, csrVal_s, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_64F);
    assert( HIPSPARSE_STATUS_SUCCESS == stat);

    //free temporary allocated variables
    err=hipFree(buffer);
    assert( hipSuccess == err );
}

void cuda_H_copy(
    cusparseSpMatDescr* &H_in,
    cusparseSpMatDescr* &H_copy){


    //declare H_in host data
    int64_t rows, cols, nnz;
    hipsparseIndexType_t csrRowOffsetsType;
    hipsparseIndexType_t csrColIndType;
    hipsparseIndexBase_t idxBase;
    hipDataType        valueType;
    //declare H_in device data
    void *csrRow, *csrCol, *csrVal;


    //get data from old sparse reference
    hipsparseStatus_t stat = hipsparseCsrGet(H_in, &rows, &cols, &nnz
                                          ,&csrRow,&csrCol, &csrVal
                                          ,&csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
    assert( HIPSPARSE_STATUS_SUCCESS == stat);

    //allocate new data arrays and copy data from previous type
    void *row_new, *col_new, *val_new;
    hipError_t err;
    err =hipMalloc( &col_new, (size_t) sizeof(int)*nnz);
    assert( hipSuccess == err );
    err = hipMemcpy(col_new, csrCol, (size_t) sizeof(int)*nnz , hipMemcpyDeviceToDevice);
    assert( hipSuccess == err );

    err =hipMalloc( &row_new, (size_t) sizeof(int)*(rows+1));
    assert( hipSuccess == err );
    err = hipMemcpy(row_new, csrRow, (size_t) sizeof(int)*(rows+1) , hipMemcpyDeviceToDevice);
    assert( hipSuccess == err );

    err =hipMalloc( &val_new, (size_t) sizeof(double)*nnz);
    assert( hipSuccess == err );
    err = hipMemcpy(val_new, csrVal, (size_t) sizeof(double)*nnz , hipMemcpyDeviceToDevice);
    assert( hipSuccess == err );

    //create new sparse matrix reference
    stat = hipsparseCreateCsr(&H_copy, rows, cols, nnz, row_new, col_new, val_new, csrRowOffsetsType, csrColIndType, idxBase, valueType);
    assert( HIPSPARSE_STATUS_SUCCESS == stat);
}


void cuda_H_destroy(
    cusparseSpMatDescr* &mat ){

    //Host data (should get destroyed with hipsparseDestroySpMat)
    int64_t rows, cols, nnz;
    hipsparseIndexType_t csrRowOffsetsType;
    hipsparseIndexType_t csrColIndType;
    hipsparseIndexBase_t idxBase;
    hipDataType        valueType;
    //Device data which must be freed first
    void *csrRow, *csrCol, *csrVal;

    //get data from old sparse reference
    hipsparseStatus_t stat = hipsparseCsrGet(mat, &rows, &cols, &nnz
                                          ,&csrRow,&csrCol, &csrVal
                                          ,&csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
    assert( HIPSPARSE_STATUS_SUCCESS == stat);

    hipError_t err;
    err=hipFree(csrRow);
    assert(err == hipSuccess);
    err=hipFree(csrCol);
    assert(err == hipSuccess);
    err=hipFree(csrVal);
    assert(err == hipSuccess);

    stat=hipsparseDestroySpMat(mat);

    assert( HIPSPARSE_STATUS_SUCCESS == stat );
}


void cuda_H_mult_r_beta(
    cusparseSpMatDescr* &mat,
    cusparseDnVecDescr* &in_vec,
    cusparseDnVecDescr* &out_vec,
    const double        &beta,
    char*               &buffer,
    hipsparseHandle_t*   &handle){

    const double alpha= 1.0;    //save in device memory
    //do the actual multiplication
    auto stat = hipsparseSpMV(*handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        mat,
                        in_vec,
                        &beta,
                        out_vec,
                        HIP_R_64F,
                        HIPSPARSE_SPMV_CSR_ALG1,
                        buffer);
#ifdef CPP_DEBUG
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
#endif
}

void cuda_H_mult_r(
    cusparseSpMatDescr* &mat,
    cusparseDnVecDescr* &in_vec,
    cusparseDnVecDescr* &out_vec,
    char*               &buffer,
    hipsparseHandle_t*   &handle){

    const double alpha= 1.0;    //save in device memory
    const double beta = 0.0;
    //do the actual multiplication
    auto stat = hipsparseSpMV(*handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        mat,
                        in_vec,
                        &beta,
                        out_vec,
                        HIP_R_64F,
                        HIPSPARSE_SPMV_CSR_ALG1,
                        buffer);
#ifdef CPP_DEBUG
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
#endif
}


void cuda_H_mult_l(
    cusparseSpMatDescr* &mat,
    cusparseDnVecDescr* &in_vec,
    cusparseDnVecDescr* &out_vec,
    char*               &buffer,
    hipsparseHandle_t*   &handle){

    const double alpha= 1.0;    //save in device memory
    const double beta = 0.0;
    //do the actual multiplication
    auto stat = hipsparseSpMV(*handle,
                        HIPSPARSE_OPERATION_TRANSPOSE,
                        &alpha,
                        mat,
                        in_vec,
                        &beta,
                        out_vec,
                        HIP_R_64F,
                        HIPSPARSE_SPMV_CSR_ALG1,
                        buffer);
#ifdef CPP_DEBUG
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
#endif
}

void cuda_H_mult_l_beta(
    cusparseSpMatDescr* &mat,
    cusparseDnVecDescr* &in_vec,
    cusparseDnVecDescr* &out_vec,
    const double        &beta,
    char*               &buffer,
    hipsparseHandle_t*   &handle){

    const double alpha= 1.0;    //save in device memory
//    const double beta = 0.0;
    //do the actual multiplication
    auto stat = hipsparseSpMV(*handle,
                        HIPSPARSE_OPERATION_TRANSPOSE,
                        &alpha,
                        mat,
                        in_vec,
                        &beta,
                        out_vec,
                        HIP_R_64F,
                        HIPSPARSE_SPMV_CSR_ALG1,
                        buffer);
#ifdef CPP_DEBUG
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
#endif
}


void cuda_free_buffer(
    char* &buffer
    ){

    auto err=hipFree(buffer);
    assert( hipSuccess == err);
}


void cuda_set_buffer(
    char* &buffer,
    cusparseSpMatDescr* &mat,
    bool transpose,
    cusparseDnVecDescr* &in_vec,
    cusparseDnVecDescr* &out_vec,
    hipsparseHandle_t* &handle){

    hipsparseStatus_t stat;
    hipError_t err;

    //get device buffer
    size_t bufferSize=0;
    const double alpha= 1.0;
    const double beta = 0.0;
    hipsparseOperation_t operation;
    if(transpose){
        operation=HIPSPARSE_OPERATION_TRANSPOSE;
    }else{
        operation=HIPSPARSE_OPERATION_NON_TRANSPOSE;
    }

    stat =hipsparseSpMV_bufferSize(*handle,
                                  operation,
                                  &alpha,
                                  mat,
                                  in_vec,
                                  &beta,
                                  out_vec,
                                  HIP_R_64F,
                                  HIPSPARSE_SPMV_CSR_ALG1,
                                  &bufferSize);
    assert (HIPSPARSE_STATUS_SUCCESS == stat);
    err=hipMalloc( (void**) &buffer, (size_t) sizeof(char)* bufferSize);
    assert( hipSuccess == err);
}

//void cuda_H_mult_mat_disc_disc(
//    //THIS ROUTINE IS NOT FINISHED AND PROBABLY NEVER WILL BE SINCE DIRECT EVALUATION OF THE SPARSE MATRICES IS FASTER DIRECTLY ON THE CPU
//    cusparseSpMatDescr* &mat,
//    cusparseDnVecDescr* &in_vec,
//    cusparseDnVecDescr* &out_vec,
//    int N_in,
//    int ind_in[],
//    double vec_in[],
//    int& N_out,  // in: size of ind_out , out: size of relevant indices in ind_out
//    int ind_out[],
//    double vec_out[]){
//
//    hipError_t err;
//
//    //host data
//    int64_t rows, cols, nnz;
//    hipsparseIndexType_t csrRowOffsetsType;
//    hipsparseIndexType_t csrColIndType;
//    hipsparseIndexBase_t idxBase;
//    hipDataType        valueType;
//    //Device data
//    void *csrRow_v, *csrCol_v, *csrVal_v;
//    int *csrRow, *csrCol;
//    double *csrVal;
//
//    //get data from old sparse reference
//    hipsparseStatus_t stat = hipsparseCsrGet(mat, &rows, &cols, &nnz
//                                          ,&csrRow_v,&csrCol_v, &csrVal_v
//                                          ,&csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
//    assert( HIPSPARSE_STATUS_SUCCESS == stat );
//
//    csrRow=(int*) csrRow_v, csrCol=(int*) csrCol_v, csrVal=(double*) csrVal_v;
//
//    //get data from input vector (only used for storage)
//    double* vec_val_dev;
//    stat=hipsparseDnVecGetValues(in_vec, (void**) &vec_val_dev);
//    assert( HIPSPARSE_STATUS_SUCCESS == stat );
//    thrust::device_ptr<double> dev_ptr(vec_val_dev);
//    thrust::fill(dev_ptr, dev_ptr + cols, 0.0);
//
////    //set in vector
////    double *in_val;
////    int *in_ind;
////    err =hipMalloc( &in_val,(size_t) sizeof(double)*N_in);
////    assert( hipSuccess == err );
////    err = hipMemcpy(in_val, vec_in, (size_t) sizeof(int)*N_in , hipMemcpyHostToDevice);
////    assert( hipSuccess == err );
////
////    err =hipMalloc( &in_ind,(size_t) sizeof(int)*N_in);
////    assert( hipSuccess == err );
////    err = hipMemcpy(in_ind, ind_in, (size_t) sizeof(int)*N_in , hipMemcpyHostToDevice);
////    assert( hipSuccess == err );
//
//
//    int N=cols;
//    int threadsPerBlock = std::min(256,N);
//    int numBlocks= (N+ threadsPerBlock - 1) / threadsPerBlock;
//
////    for (int i =0; i<N_in; ++i){
////        add_col_mat<<<numBlocks, threadsPerBlock>>>(vec_val_dev,csrRow, csrCol, csrVal, i, ind_in[i], vec_in[i]);
////    }
//}

}
